#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define THREADS_PER_BLOCK 512
#define BLOCKS_PER_GRID 12

#define N 1000000
#define width 1.0 / N



__global__ void parallel_pi(double *sum){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int id = index;

    while( id < N){ 
        //Variables declared to keep up with every thread
        double mid = (id + 0.5) * width;
        double height = 4.0 / (1.0 + mid * mid);
        sum[index] += height;
        id = id + blockDim.x * gridDim.x;
    }
}

void sequential_pi(){
    // Code given in the Cuda Material
    long num_rects = 100000, i;
    double mid, height, w, area;
    double sum = 0.0;
    w = 1.0 / (double) num_rects;
    for (i = 0; i < num_rects; i++) {
        mid = (i + 0.5) * w;
        height = 4.0 / (1.0 + mid * mid);
        sum += height;
   }
   area = w * sum;
   printf("Sequential Pi: %f\n",area);
}

int main(){
    double area = 0;
    double *d_s;
    double *sum;
    int size = THREADS_PER_BLOCK * BLOCKS_PER_GRID;
    
    
    sum = (double*) malloc(size * sizeof(double));

    hipMalloc((void**) &d_s, size * sizeof(double));

    parallel_pi<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(d_s);

    hipMemcpy(sum, d_s, size * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++) {
        area += sum[i];
    }
    area = area * width;

    printf("Multithread Pi: %f\n", area);
    sequential_pi();


    free(sum);
    hipFree(d_s);



    return 0;
}